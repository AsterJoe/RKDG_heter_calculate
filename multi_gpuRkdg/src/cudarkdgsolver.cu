#include "hip/hip_runtime.h"
#include "../inc/cudarkdgsolver.h"

CCUDARkdgSolver::CCUDARkdgSolver():
num_commu(0),
title("Unknown case"),
alpha(5.0),
gamma(1.4),
mach(0.4),
cfl(0.18),
rhoref(1.0),
pref(1.0),

_terminal_time(15),
log_history('Y'),
print_interval(1000),

gridconf("input/mesh.conf"),
solution_file("output/solution.dat"),
log_file("output/log.dat"),
residual_file("output/residual.dat"),

threads_per_block(512),
reduction_threads(512),

_freedom_rho(NULL),
_freedom_rhou(NULL),
_freedom_rhov(NULL),
_freedom_rhoE(NULL),

_dt(NULL),
_residual(NULL)
{}


void CCUDARkdgSolver::detectCUDADevice( void )
{
	int count(0);

	hipGetDeviceCount( &count );
	cout<<"count:"<<count;
	/*if ( 0==count )
		throw CMyException("No device surpports CUDA found!");*/
	//if ( count < nprocs) {
	//	throw CMyException("No enough device surpports CUDA found!");
	//}

	hipDeviceProp_t prop;

	bool double_support(false);

	/*for ( int i=0; i<count; ++i )
	{
		hipGetDeviceProperties( &prop, i );
		if ( prop.major>1 )
		{
			double_support = true;
			break;
		}
	}

	if ( !double_support )
		throw CMyException("No device has capability of 2.0 or higher is found!");*/

	int double_support_count(0);
	for ( int i=0; i<count; ++i )
	{
		hipGetDeviceProperties( &prop, i );
		if ( prop.major>1 )
		{
			double_support_count++;
			if (double_support_count == myid) {
				hipSetDevice(i);
			}
		}
		cout<<"gpu("<<i<<"):"<<prop.major<<"."<<prop.minor<<endl;
	}

//	if ( double_support_count < nprocs )
//		throw CMyException("No enough device has capability of 2.0 or higher is found!");

	/*memset( &prop, 0, sizeof(hipDeviceProp_t) );
	prop.major = 2;
	prop.minor = 0;

	int devid;


	hipChooseDevice(&devid, &prop);*/

	//cout<<"\nThere are "<<count<<" device surpports CUDA, and the "<<devid+1<<"th device will be used."<<endl;
}

void CCUDARkdgSolver::initConfig(void)
{
	string conf_items[] = {
		"title",
		"gamma", "alpha", "mach", "cfl", "rhoref","pref",
		"time",  
		"gridconf", "logfile",  "solutionfile", "residualfile", 
		"threadsperblock", "reductionthreads",
		"loghistory", "printinterval"
	};

	CConfig program_conf(config_file, conf_items, 16);

	program_conf.parseConfigFile();

	// ת�����ò���
	if ( program_conf.config_items["title"]!="" )
		title = program_conf.config_items["title"];

	if ( program_conf.config_items["gamma"]!="" )
		gamma = atof(program_conf.config_items["gamma"].c_str());
		
	if ( program_conf.config_items["alpha"]!="" )
		alpha = atof(program_conf.config_items["alpha"].c_str())*atan(1.0)*4 / 180;
		
	if ( program_conf.config_items["mach"]!="" )
		mach  = atof(program_conf.config_items["mach"].c_str());
	
	if ( program_conf.config_items["cfl"]!="" )
		cfl   = atof(program_conf.config_items["cfl"].c_str());
		
	if ( program_conf.config_items["rhoref"]!="" )
		rhoref = atof(program_conf.config_items["rhoref"].c_str());
	
	if ( program_conf.config_items["pref"]!="" )
		pref = atof(program_conf.config_items["pref"].c_str());

	if ( program_conf.config_items["time"]!="" )
		_terminal_time = atof(program_conf.config_items["time"].c_str());

	if ( program_conf.config_items["gridconf"]!="" )
		gridconf = program_conf.config_items["gridconf"];

	if ( program_conf.config_items["solutionfile"]!="" )
		solution_file = program_conf.config_items["solutionfile"];
		
	if ( program_conf.config_items["logfile"]!="" )
		log_file = program_conf.config_items["logfile"];

	if ( program_conf.config_items["residualfile"]!="" )
		residual_file = program_conf.config_items["residualfile"];
		
	if ( program_conf.config_items["loghistory"]!="" )
		log_history = toupper(program_conf.config_items["loghistory"].at(0));
		
	if ( program_conf.config_items["threadsperblock"]!="" )	
		threads_per_block = atoi(program_conf.config_items["threadsperblock"].c_str());

	if ( program_conf.config_items["printinterval"]!="" )	
		print_interval = abs(atoi(program_conf.config_items["printinterval"].c_str()));
	
	if ( program_conf.config_items["reductionthreads"]!="" )
		reduction_threads = atoi(program_conf.config_items["reductionthreads"].c_str());
}

void CCUDARkdgSolver::run(int myid, int nprocs)
{
	this->myid = myid;
	this->nprocs = nprocs;
	ofstream fout(log_file.c_str());
	if ( !fout )
		throw CMyException("Failed to open log file: "+log_file);
	
	CMyTime mt;
	
	
	fout<<mt.getCurrentTime()<<": programs starts"<<endl;

	// ���CUDA�豸
	detectCUDADevice();
	fout<<mt.getCurrentTime()<<": Device with capability of 2.0 is found."<<endl;

	
	// ��ʼ���������ò������������
	initConfig();
	fout<<mt.getCurrentTime()<<": initialize configure from file."<<endl<<endl;
	fout<<"Title: "<<title<<endl<<endl;
	printConfig(cout);
	printConfig(fout);
	
	
	fout<<mt.getCurrentTime()<<": reading grid information."<<endl;

	// ��ʼ������
	grid.config_file = gridconf;
	grid.initializeGrid(myid, nprocs);
	fout.close();
}

void CCUDARkdgSolver::runNext() {
	cout<<"runNext"<<endl;
	grid.initializeGridNext();
	grid.outputGrid();
	grid.outputGridWithGhostCells("output/ghostmesh.plt");
	
	//fout<<mt.getCurrentTime()<<": complete grid initialization."<<endl;

	// ���������ζ����Ƿ���ʱ������
//	grid.testTrianglesAntiwise();
	grid.testLocalTriangleAntiwise();

	// ��ʼ�������ϻ���������Ϣ
//	grid.triangle_infos.allocateMemory(grid.getCellNumber());
	grid.triangle_infos.allocateMemory(grid.getLocalCellNumber());
	grid.initializeTriangleInfos();

//	fout<<mt.getCurrentTime()<<": complete grid information initialization."<<endl;

	// �������Ԫ
//	grid.markBoundaryTriangles();
	grid.markLocalBoundaryTriangles();
	/*for (int i = 0; i < grid.getLocalCellNumber(); i++) {
		cout<<"flag"<<grid.area_index<<":"<<grid.local_tri_flag[i]<<endl;
	}*/
	// ����GPU�ڴ�
//	_cuarrays.allocateMemory(grid.getCellNumber());

	_cuarrays.allocateMemory(grid.getLocalCellNumber());
	cout<<"ab"<<endl;
	// �����ǵ�Ԫ��Ϣ���͵�GPU
	copyTriangleInfosToGPU();
	cout<<"cd"<<endl;
	// ��ʼ��RKDG���ɶȣ�������ʼ�����ݴ���GPU
	initRKDG();
	cout<<"ef"<<endl;
//	fout<<mt.getCurrentTime()<<": program initialization complete."<<endl;

//	fout<<mt.getCurrentTime()<<": begin to solve flow."<<endl<<endl;
	/** ʱ���ƽ�*/
	//mt.beginTimer();
	//rkdgAdvance();
//	fout.close();
}

void CCUDARkdgSolver::runAfter()
{
	//mt.endTimer();
	
	//fout<<"RKDG performance:"<<endl;
	//fout<<"CPU time:  "<<mt.getCPUElapsedTime()<<" s"<<endl;
	//fout<<"wall time: "<<mt.getWallElapsedTime()<<" s"<<endl<<endl;
	
	//fout<<mt.getCurrentTime()<<": complete solving flow."<<endl;

	// �������ɶȵ�����
	copyFreedomToHost();
	
	// �����
	//outputSolution();

	//fout<<mt.getCurrentTime()<<": complete solution output."<<endl;
	
	//fout.close();
}

void CCUDARkdgSolver::copyFreedomToHost()
{
	size_t size = sizeof(double)*grid.getLocalCellNumber();
	size_t pitch = _cuarrays.getDoublePitch();

	hipMemcpy2D(_freedom_rho,  size, _cuarrays.freedom_rho,  pitch, size, BASIS_FUNCTIONS, hipMemcpyDeviceToHost);			  
	hipMemcpy2D(_freedom_rhou, size, _cuarrays.freedom_rhou, pitch, size, BASIS_FUNCTIONS, hipMemcpyDeviceToHost);			  
	hipMemcpy2D(_freedom_rhov, size, _cuarrays.freedom_rhov, pitch, size, BASIS_FUNCTIONS, hipMemcpyDeviceToHost);			 
	hipMemcpy2D(_freedom_rhoE, size, _cuarrays.freedom_rhoE, pitch, size, BASIS_FUNCTIONS, hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
}

void CCUDARkdgSolver::copyTriangleInfosToGPU(void)
{
//	int num = grid.getCellNumber();
	int num = grid.getLocalCellNumber();
	size_t int_pitch    = _cuarrays.getIntPitch();
	size_t double_pitch = _cuarrays.getDoublePitch();

//	hipMemcpy2DAsync(_cuarrays.neighbour, int_pitch, grid.tri_neighbour, sizeof(int)*num, sizeof(int)*num, TRIANGLE_EDGES, hipMemcpyHostToDevice);
	hipMemcpy2DAsync(_cuarrays.neighbour, int_pitch, grid.local_tri_neighbour, sizeof(int)*num, sizeof(int)*num, TRIANGLE_EDGES, hipMemcpyHostToDevice);

//	hipMemcpy2DAsync(_cuarrays.sharedEdge, int_pitch, grid.tri_sharedEdge, sizeof(int)*num, sizeof(int)*num, TRIANGLE_EDGES, hipMemcpyHostToDevice);
	hipMemcpy2DAsync(_cuarrays.sharedEdge, int_pitch, grid.local_tri_sharedEdge, sizeof(int)*num, sizeof(int)*num, TRIANGLE_EDGES, hipMemcpyHostToDevice);

//	hipMemcpy2DAsync(_cuarrays.triangle_flag, int_pitch, grid.tri_flag, sizeof(int)*num, sizeof(int)*num, 1, hipMemcpyHostToDevice);
	hipMemcpy2DAsync(_cuarrays.triangle_flag, int_pitch, grid.local_tri_flag, sizeof(int)*num, sizeof(int)*num, 1, hipMemcpyHostToDevice);

	if ( hipPeekAtLastError() !=hipSuccess )
	{
		cout<<"throw error previous!"<<endl;
	} else {
		cout<<"not throw error previous!"<<endl;
	}

	size_t gsize = sizeof(double)*num;

	/*hipMemcpy2DAsync(_cuarrays.area, double_pitch, grid.triangle_infos.area, gsize, gsize, 1, hipMemcpyHostToDevice);
	
	hipMemcpy2DAsync(_cuarrays.perimeter, double_pitch, grid.triangle_infos.perimeter, gsize, gsize, 1, hipMemcpyHostToDevice);

	hipMemcpy2DAsync(_cuarrays.outer_normal_vector, double_pitch, grid.triangle_infos.outer_normal_vector, gsize, gsize, TRIANGLE_EDGES*2, hipMemcpyHostToDevice);

	hipMemcpy2DAsync(_cuarrays.mass_coeff, double_pitch, grid.triangle_infos.mass_coeff, gsize, gsize, BASIS_FUNCTIONS, hipMemcpyHostToDevice);

	hipMemcpy2DAsync(_cuarrays.vol_bf_value, double_pitch, grid.triangle_infos.vol_bf_value, gsize, gsize, VOLUME_GPOINTS*BASIS_FUNCTIONS, hipMemcpyHostToDevice);

	hipMemcpy2DAsync(_cuarrays.vol_bdf_value, double_pitch, grid.triangle_infos.vol_bdf_value, gsize, gsize, VOLUME_GPOINTS*BASIS_FUNCTIONS*2, hipMemcpyHostToDevice);

	hipMemcpy2DAsync(_cuarrays.edge_bf_value, double_pitch, grid.triangle_infos.edge_bf_value, gsize, gsize, TRIANGLE_EDGES*EDGE_GPOINTS*BASIS_FUNCTIONS, hipMemcpyHostToDevice);

	hipMemcpy2DAsync(_cuarrays.vol_gauss_weight, double_pitch, grid.triangle_infos.vol_gauss_weight, gsize, gsize,  VOLUME_GPOINTS, hipMemcpyHostToDevice);

	hipMemcpy2DAsync(_cuarrays.edge_gauss_weight, double_pitch, grid.triangle_infos.edge_gauss_weight, gsize, gsize, EDGE_GPOINTS*TRIANGLE_EDGES, hipMemcpyHostToDevice);*/

	hipMemcpy2DAsync(_cuarrays.area, double_pitch, grid.triangle_infos.area, gsize, gsize, 1, hipMemcpyHostToDevice);

	hipMemcpy2DAsync(_cuarrays.perimeter, double_pitch, grid.triangle_infos.perimeter, gsize, gsize, 1, hipMemcpyHostToDevice);

	hipMemcpy2DAsync(_cuarrays.outer_normal_vector, double_pitch, grid.triangle_infos.outer_normal_vector, gsize, gsize, TRIANGLE_EDGES*2, hipMemcpyHostToDevice);

	hipMemcpy2DAsync(_cuarrays.mass_coeff, double_pitch, grid.triangle_infos.mass_coeff, gsize, gsize, BASIS_FUNCTIONS, hipMemcpyHostToDevice);

	hipMemcpy2DAsync(_cuarrays.vol_bf_value, double_pitch, grid.triangle_infos.vol_bf_value, gsize, gsize, VOLUME_GPOINTS*BASIS_FUNCTIONS, hipMemcpyHostToDevice);

	hipMemcpy2DAsync(_cuarrays.vol_bdf_value, double_pitch, grid.triangle_infos.vol_bdf_value, gsize, gsize, VOLUME_GPOINTS*BASIS_FUNCTIONS*2, hipMemcpyHostToDevice);

	hipMemcpy2DAsync(_cuarrays.edge_bf_value, double_pitch, grid.triangle_infos.edge_bf_value, gsize, gsize, TRIANGLE_EDGES*EDGE_GPOINTS*BASIS_FUNCTIONS, hipMemcpyHostToDevice);

	hipMemcpy2DAsync(_cuarrays.vol_gauss_weight, double_pitch, grid.triangle_infos.vol_gauss_weight, gsize, gsize,  VOLUME_GPOINTS, hipMemcpyHostToDevice);

	hipMemcpy2DAsync(_cuarrays.edge_gauss_weight, double_pitch, grid.triangle_infos.edge_gauss_weight, gsize, gsize, EDGE_GPOINTS*TRIANGLE_EDGES, hipMemcpyHostToDevice);

	if ( hipPeekAtLastError()!=hipSuccess )
	{
		throw CMyException(hipGetErrorString(hipPeekAtLastError()));
	}
}

void CCUDARkdgSolver::initRKDG()
{
//	int num = grid.getCellNumber();
	int num = grid.getLocalCellNumber();
	double  ut = sqrt(gamma*pref/rhoref)*mach;

	double u = ut * cos(alpha);
	double v = ut * sin(alpha);

	// �����ڴ�
	_freedom_rho  = new double[num*BASIS_FUNCTIONS];
	_freedom_rhou = new double[num*BASIS_FUNCTIONS];
	_freedom_rhov = new double[num*BASIS_FUNCTIONS];
	_freedom_rhoE = new double[num*BASIS_FUNCTIONS];

	hipHostAlloc((void**)&_dt, sizeof(double), hipHostMallocDefault);
	hipHostAlloc((void**)&_residual, sizeof(double)*RESIDUAL_VARS, hipHostMallocDefault);

	if ( hipPeekAtLastError()!=hipSuccess )
		throw CMyException(hipGetErrorString(hipPeekAtLastError()));

	// ��ʼ�����ɶȵ�ֵ
	for ( int i=0; i<num; ++i )
	{
		_freedom_rho[i] = rhoref;
		_freedom_rhou[i] = rhoref *u;
		_freedom_rhov[i] = rhoref*v;
		_freedom_rhoE[i] = rhoref*(ut*ut)/2 + pref/(rhoref*(gamma-1));
	}

	int dev_pitch = _cuarrays.getDoublePitch();
	int host_pitch = sizeof(double)*num;
	hipMemsetAsync(_cuarrays.freedom_rho,  0, dev_pitch*BASIS_FUNCTIONS);
	hipMemsetAsync(_cuarrays.freedom_rhou, 0, dev_pitch*BASIS_FUNCTIONS);
	hipMemsetAsync(_cuarrays.freedom_rhov, 0, dev_pitch*BASIS_FUNCTIONS);
	hipMemsetAsync(_cuarrays.freedom_rhoE, 0, dev_pitch*BASIS_FUNCTIONS);

	hipMemcpy2DAsync(_cuarrays.freedom_rho,  dev_pitch, _freedom_rho,  host_pitch, host_pitch, 1, hipMemcpyHostToDevice);														   
	hipMemcpy2DAsync(_cuarrays.freedom_rhou, dev_pitch, _freedom_rhou, host_pitch, host_pitch, 1, hipMemcpyHostToDevice);														   
	hipMemcpy2DAsync(_cuarrays.freedom_rhov, dev_pitch, _freedom_rhov, host_pitch, host_pitch, 1, hipMemcpyHostToDevice);														   
	hipMemcpy2DAsync(_cuarrays.freedom_rhoE, dev_pitch, _freedom_rhoE, host_pitch, host_pitch, 1, hipMemcpyHostToDevice);
}

void CCUDARkdgSolver::getTimeStep(int tnum)
{
	kernel_getTimeStep<<<1,reduction_threads, sizeof(double)*reduction_threads>>>(
		tnum, gamma, cfl, _cuarrays.ddt,

		_cuarrays.freedom_rho,  _cuarrays.freedom_rhou,
		_cuarrays.freedom_rhov, _cuarrays.freedom_rhoE,
		
		_cuarrays.perimeter, _cuarrays.area
		);
}

void CCUDARkdgSolver::calculateConVars(int tnum, int double_pitch, int blocks)
{
	size_t size = sizeof(double)*threads_per_block*CONSERVATIVE_VARS;

	kernel_calculateConVars<<<blocks,threads_per_block, size>>>(
		tnum, double_pitch,
		_cuarrays.freedom_rho,   _cuarrays.freedom_rhou,
		_cuarrays.freedom_rhov,  _cuarrays.freedom_rhoE,

		_cuarrays.convar_rho_vol,   _cuarrays.convar_rhou_vol,
		_cuarrays.convar_rhov_vol,  _cuarrays.convar_rhoE_vol,

		_cuarrays.convar_rho_edge,  _cuarrays.convar_rhou_edge, 
		_cuarrays.convar_rhov_edge, _cuarrays.convar_rhoE_edge, 

		_cuarrays.vol_bf_value,  _cuarrays.edge_bf_value
		);
}

void CCUDARkdgSolver::boundaryCondition(int tnum, int num, int double_pitch, double rho, double rhou, double rhov, double rhoE)
{
	// �߽絥Ԫ����Ŀ���٣�Ϊ��������ܣ���ÿ��block����ɼ����߳������������
	int threads = 64;

	int blocks = ((num-tnum)%threads) ? (num-tnum)/threads+1 : (num-tnum)/threads;

	kernel_boundaryCondition<<<blocks,threads>>>(
		tnum, num, double_pitch,
		rho, rhou, rhov, rhoE,

		_cuarrays.convar_rho_edge,   _cuarrays.convar_rhou_edge,
		_cuarrays.convar_rhov_edge,  _cuarrays.convar_rhoE_edge,

		_cuarrays.freedom_rho,   _cuarrays.freedom_rhou,
		_cuarrays.freedom_rhov,  _cuarrays.freedom_rhoE,

		_cuarrays.neighbour,     _cuarrays.sharedEdge,
		  
		_cuarrays.triangle_flag, _cuarrays.outer_normal_vector
		);
}

void CCUDARkdgSolver::calculateVolumeRHS(int tnum, int double_pitch, int blocks)
{
	size_t size = sizeof(double)*threads_per_block*VOLUME_GPOINTS;
	
	kernel_calculateVolumeRHS<<<blocks, threads_per_block, size>>>(
		tnum, double_pitch, gamma, 

		_cuarrays.convar_rho_vol, _cuarrays.convar_rhou_vol, 
		_cuarrays.convar_rhov_vol, _cuarrays.convar_rhoE_vol,

		_cuarrays.rhs_volume_rho, _cuarrays.rhs_volume_rhou, 
		_cuarrays.rhs_volume_rhov, _cuarrays.rhs_volume_rhoE,

		_cuarrays.vol_gauss_weight, _cuarrays.vol_bdf_value
		);
}

void CCUDARkdgSolver::calculateLFCoeff(int tnum, int ipitch_num, int dpitch_num, int blocks)
{

	kernel_calculateLFCoeff<<<blocks, threads_per_block>>>(
		tnum, ipitch_num, dpitch_num, gamma, 
		_cuarrays.outer_normal_vector,  _cuarrays.neighbour, 

		_cuarrays.freedom_rho, _cuarrays.freedom_rhou, 
		_cuarrays.freedom_rhov, _cuarrays.freedom_rhoE, 

		_cuarrays.lfflux_coeff
		);
}

void CCUDARkdgSolver::calculateEdgeFG(int tnum, int num, int double_pitch, int blocks)
{
	// �˴���Ҫ����ĵ�Ԫ������������һ�����Ӷ��߳̿���Ҫ���¶���
	blocks = (num%threads_per_block) ? num/threads_per_block+1 : num/threads_per_block;
	
	kernel_calculateEdgeFG<<<blocks, threads_per_block>>>(
		tnum, num, double_pitch, gamma, 

		_cuarrays.convar_rho_edge,  _cuarrays.convar_rhou_edge,
		_cuarrays.convar_rhov_edge, _cuarrays.convar_rhoE_edge,

		_cuarrays.fedge_rho,  _cuarrays.fedge_rhou,
		_cuarrays.fedge_rhov, _cuarrays.fedge_rhoE,

		_cuarrays.gedge_rho,  _cuarrays.gedge_rhou,
		_cuarrays.gedge_rhov, _cuarrays.gedge_rhoE
		);
}

void CCUDARkdgSolver::calculateFlux(int tnum, int int_pitch, int double_pitch, int blocks)
{
/*
	kernel_calculateFlux<<<blocks, threads_per_block>>>(
		tnum, int_pitch, double_pitch,
		_cuarrays.neighbour, _cuarrays.sharedEdge,

		_cuarrays.convar_rho_edge,  _cuarrays.convar_rhou_edge,
		_cuarrays.convar_rhov_edge, _cuarrays.convar_rhoE_edge,

		_cuarrays.fedge_rho,  _cuarrays.fedge_rhou,
		_cuarrays.fedge_rhov, _cuarrays.fedge_rhoE,

		_cuarrays.gedge_rho,  _cuarrays.gedge_rhou,
		_cuarrays.gedge_rhov, _cuarrays.gedge_rhoE,

		_cuarrays.outer_normal_vector, _cuarrays.lfflux_coeff,

		_cuarrays.lfflux_rho,  _cuarrays.lfflux_rhou,
		_cuarrays.lfflux_rhov, _cuarrays.lfflux_rhoE
		);
*/
	kernel_calculateFlux<<<blocks, threads_per_block>>>(
		tnum, int_pitch, double_pitch,
		_cuarrays.neighbour, _cuarrays.sharedEdge,

		_cuarrays.convar_rho_edge,  _cuarrays.convar_rhou_edge,
//		_cuarrays.convar_rhov_edge, _cuarrays.convar_rhoE_edge,

		_cuarrays.fedge_rho,  _cuarrays.fedge_rhou,
//		_cuarrays.fedge_rhov, _cuarrays.fedge_rhoE,

		_cuarrays.gedge_rho,  _cuarrays.gedge_rhou,
//		_cuarrays.gedge_rhov, _cuarrays.gedge_rhoE,

		_cuarrays.outer_normal_vector, _cuarrays.lfflux_coeff,

		_cuarrays.lfflux_rho,  _cuarrays.lfflux_rhou
//		_cuarrays.lfflux_rhov, _cuarrays.lfflux_rhoE
		);
		
	kernel_calculateFlux<<<blocks, threads_per_block>>>(
		tnum, int_pitch, double_pitch,
		_cuarrays.neighbour, _cuarrays.sharedEdge,

//		_cuarrays.convar_rho_edge,  _cuarrays.convar_rhou_edge,
		_cuarrays.convar_rhov_edge, _cuarrays.convar_rhoE_edge,

//		_cuarrays.fedge_rho,  _cuarrays.fedge_rhou,
		_cuarrays.fedge_rhov, _cuarrays.fedge_rhoE,

//		_cuarrays.gedge_rho,  _cuarrays.gedge_rhou,
		_cuarrays.gedge_rhov, _cuarrays.gedge_rhoE,

		_cuarrays.outer_normal_vector, _cuarrays.lfflux_coeff,

//		_cuarrays.lfflux_rho,  _cuarrays.lfflux_rhou,
		_cuarrays.lfflux_rhov, _cuarrays.lfflux_rhoE
		);
}

void CCUDARkdgSolver::calculateEdgeRHS(int tnum, int double_pitch, int blocks)
{
	size_t size = sizeof(double)*threads_per_block*TRIANGLE_EDGES*EDGE_GPOINTS;

	kernel_calculateEdgeRHS<<<blocks, threads_per_block, size>>>(
		tnum, double_pitch,
		_cuarrays.edge_gauss_weight, _cuarrays.edge_bf_value,

		_cuarrays.lfflux_rho,  _cuarrays.lfflux_rhou, 
		_cuarrays.lfflux_rhov, _cuarrays.lfflux_rhoE, 

		_cuarrays.rhs_edge_rho,  _cuarrays.rhs_edge_rhou, 
		_cuarrays.rhs_edge_rhov, _cuarrays.rhs_edge_rhoE,

		_cuarrays.rhs_volume_rho,  _cuarrays.rhs_volume_rhou,
		_cuarrays.rhs_volume_rhov, _cuarrays.rhs_volume_rhoE
		);

}

void CCUDARkdgSolver::rkdgStepOne(double dt, int tnum, int double_pitch, int blocks)
{

	kernel_rkdgStepOne<<<blocks, threads_per_block>>>(
		tnum, double_pitch, dt, _cuarrays.mass_coeff,

		_cuarrays.freedom_rho,  _cuarrays.freedom_rhou, 
		_cuarrays.freedom_rhov, _cuarrays.freedom_rhoE, 

		_cuarrays.rhs_edge_rho,  _cuarrays.rhs_edge_rhou,
		_cuarrays.rhs_edge_rhov, _cuarrays.rhs_edge_rhoE
		);
}

void CCUDARkdgSolver::rkdgStepTwo(double dt, int tnum, int double_pitch, int blocks)
{
	kernel_rkdgStepTwo<<<blocks, threads_per_block>>>(
		tnum, double_pitch, dt, _cuarrays.mass_coeff,
		_cuarrays.freedom_rho,  _cuarrays.freedom_rhou, 
		_cuarrays.freedom_rhov, _cuarrays.freedom_rhoE, 

		_cuarrays.rhs_edge_rho,  _cuarrays.rhs_edge_rhou,
		_cuarrays.rhs_edge_rhov, _cuarrays.rhs_edge_rhoE,

		_cuarrays.freedom_rho_old,  _cuarrays.freedom_rhou_old, 
		_cuarrays.freedom_rhov_old, _cuarrays.freedom_rhoE_old
		);
}

void CCUDARkdgSolver::rkdgStepThree(double dt, int tnum, int double_pitch, int blocks)
{

	kernel_rkdgStepThree<<<blocks, threads_per_block>>>(
		tnum, double_pitch, dt, _cuarrays.mass_coeff,

		_cuarrays.freedom_rho,     _cuarrays.freedom_rhou, 
		_cuarrays.freedom_rhov,    _cuarrays.freedom_rhoE, 

		_cuarrays.rhs_edge_rho,    _cuarrays.rhs_edge_rhou,
		_cuarrays.rhs_edge_rhov,   _cuarrays.rhs_edge_rhoE,

		_cuarrays.freedom_rho_old,     _cuarrays.freedom_rhou_old, 
		_cuarrays.freedom_rhov_old,    _cuarrays.freedom_rhoE_old
		);
}

void CCUDARkdgSolver::calculateResidual(int tnum)
{
	kernel_calculateResidual<<<1,reduction_threads, sizeof(double)*reduction_threads*RESIDUAL_VARS>>>(
		tnum, 
		_cuarrays.freedom_rho, _cuarrays.freedom_rhoE,

		_cuarrays.freedom_rho_old, _cuarrays.freedom_rhoE_old,

		_cuarrays.residual
		);
}

//void CCUDARkdgSolver::rkdgAdvance(void)
//{
//	ofstream fout;
//	if ( log_history=='Y' )
//	{
//		fout.open(residual_file.c_str());
//		if ( !fout )
//			throw CMyException("Failed to open residual log file: "+residual_file);
//
//		fout<<"N, rho"<<endl;
//	}
//	
//	
//	double nt(0);
//	int count(0);
//
//	/*int tnum = grid.getTriangleNumber();
//	int num  = grid.getCellNumber();*/
//	int tnum = grid.getLocalTriangleNumber();
//	int num = grid.getLocalCellNumber();
//
//
//	int blocks = (tnum%threads_per_block) ? tnum/threads_per_block+1 : tnum/threads_per_block;
//
//	double ut   = sqrt(gamma*pref/rhoref)*mach;
//	double rhou = rhoref*ut*cos(alpha);
//	double rhov = rhoref*ut*sin(alpha);
//	double rhoE = 0.5*rhoref*(ut*ut) + pref/(gamma-1);
//
//	bool copy(false);
//
//	hipError_t error;
//	size_t pitch = _cuarrays.getDoublePitch();
//	int pitch_num = pitch / sizeof(double);
//	int ipitch_num = _cuarrays.getIntPitch() / sizeof(int);
//
//	hipEvent_t time_start, time_stop;
//
//	hipEventCreateWithFlags(&time_start, hipEventDisableTiming|hipEventBlockingSync);
//	hipEventCreateWithFlags(&time_stop,  hipEventDisableTiming|hipEventBlockingSync);
//
//	if ( log_history=='Y' )
//		copy = true;
//
//	// ȷ��֮ǰCUDA�ĳ�ʼ���������Ѿ����
//	hipDeviceSynchronize();
//
//	do 
//	{
//		++ count;
//		
//		hipEventRecord(time_start);
//		// ���㵱ǰʱ�䲽��
//		getTimeStep(tnum);
//
//		hipEventRecord(time_stop);
//
//		// ��������ɶ�
//		/*hipMemcpy2DAsync(_cuarrays.freedom_rho_old,  pitch, _cuarrays.freedom_rho,  pitch, pitch, BASIS_FUNCTIONS, hipMemcpyDeviceToDevice);							  
//		hipMemcpy2DAsync(_cuarrays.freedom_rhou_old, pitch, _cuarrays.freedom_rhou, pitch, pitch, BASIS_FUNCTIONS, hipMemcpyDeviceToDevice);							  
//		hipMemcpy2DAsync(_cuarrays.freedom_rhov_old, pitch, _cuarrays.freedom_rhov, pitch, pitch, BASIS_FUNCTIONS, hipMemcpyDeviceToDevice);							 
//		hipMemcpy2DAsync(_cuarrays.freedom_rhoE_old, pitch, _cuarrays.freedom_rhoE, pitch, pitch, BASIS_FUNCTIONS, hipMemcpyDeviceToDevice);*/
//			
//		hipMemcpy2DAsync(_freedom_rho, pitch, _cuarrays.freedom_rho, pitch, pitch, BASIS_FUNCTIONS, hipMemcpyDeviceToHost);
//		hipMemcpy2DAsync(_freedom_rho, pitch, _cuarrays.freedom_rho, pitch, pitch, BASIS_FUNCTIONS, hipMemcpyDeviceToHost);
//		hipMemcpy2DAsync(_freedom_rho, pitch, _cuarrays.freedom_rho, pitch, pitch, BASIS_FUNCTIONS, hipMemcpyDeviceToHost);
//		hipMemcpy2DAsync(_freedom_rho, pitch, _cuarrays.freedom_rho, pitch, pitch, BASIS_FUNCTIONS, hipMemcpyDeviceToHost);
//
//		
//
//		MPI_Request request[4], request1;
//		MPI_Status status;
//		int x = 1;         //֪ͨ�����ڵ���Ϣ������0��ʾ����
//		if(myid = 0 && count % 5 == 0) {
//			for (int i = 1; i < nprocs; i++) {
//				MPI_Isend(&x, 1, MPI_INT, i, 0, MPI_COMM_WORLD, &request1);
//			}
//			commuInfo();
//		} else if(myid != 0) {
//			int flag;
//			MPI_Status status;
//			if (count == 0) {
//				MPI_Irecv(&x,1, MPI_INT, 0, 0, MPI_COMM_WORLD, &request1);
//			} 
//			MPI_Test(&request1, &flag, &status);
//			if (flag == 1 && x == 1) {
//				commuInfo();
//				MPI_Irecv(&x,1, MPI_INT, 0, 0, MPI_COMM_WORLD, &request1);
//			}
//		}
//
//		
//		for ( int i=0; i<RUNGE_KUTTA_STEPS; ++i )
//		{
//			// �����غ�����ֵ
//			calculateConVars(tnum, pitch_num, blocks);
//
//			// ����߽�����
//			boundaryCondition(tnum, num, pitch_num, rhoref, rhou, rhov, rhoE);
//
//			// ��������ֲв�
//			calculateVolumeRHS(tnum, pitch_num, blocks);
//
//			// ����LFͨ��ϵ��
//			calculateLFCoeff(tnum, ipitch_num, pitch_num, blocks);
//
//			// ����f, g�ڱ��ϵ�ֵ
//			calculateEdgeFG(tnum, num, pitch_num, blocks);
//
//			calculateFlux(tnum, ipitch_num, pitch_num, blocks);
//
//			// �����߻��ֲв�
//			calculateEdgeRHS(tnum, pitch_num, blocks);
//
//			// ʱ���ƽ�
//			switch (i)
//			{
//			case 0:
//				hipEventSynchronize(time_stop);
//
//				// ��ʱ�䲽�����͵�����
//				hipMemcpy(_dt, _cuarrays.ddt, sizeof(double), hipMemcpyDeviceToHost);
//
//				if ( 0==(count-1)%print_interval )
//					cout<<"Step: "<<count<<", time step: "<<_dt[0]<<endl;
//
//				if ( (_terminal_time-nt)<_dt[0] )
//				{
//					_dt[0] = _terminal_time -  nt;
//				}
//
//				// ʱ�䲽�ƽ�
//				rkdgStepOne(_dt[0], tnum, pitch_num, blocks);
//
//				break;
//
//			case 1:
//				rkdgStepTwo(_dt[0], tnum, pitch_num, blocks);
//				break;
//
//			case 2:
//				rkdgStepThree(_dt[0], tnum, pitch_num, blocks);
//				break;
//
//			default:
//				throw CMyException("impossible case!");
//				break;
//			}
//		}
//
//		
//		if ( copy && (count-1) )
//		{
//			// ���Ʋв�����
//			hipMemcpy(_residual, _cuarrays.residual,
//						sizeof(double)*RESIDUAL_VARS, hipMemcpyDeviceToHost);
//
//			if ( 0==(count-1)%print_interval )
//				cout<<"Current time: "<<nt<<"   rhomax: "<<_residual[0]/rhoref<<"   E: "<<_residual[1]/rhoE<<endl;
//
//			fout<<count<<"   "<<log(_residual[0]/rhoref)/log(10.0)<<endl;
//		}
//
//		// ����в�
//		calculateResidual(tnum);
//
//		
//		// ��ʱ�ƽ�
//		nt += _dt[0];
//
//		error = hipPeekAtLastError();
//		if ( error!=hipSuccess )
//			throw CMyException(hipGetErrorString(error));
//
//	} while ( nt<_terminal_time );
//
//	hipDeviceSynchronize();
//	
//	if ( copy )
//	{
//		// ���Ʋв�����
//		hipMemcpy(_residual, _cuarrays.residual,
//			sizeof(double)*RESIDUAL_VARS, hipMemcpyDeviceToHost);
//
//		if ( 0==(count-1)%print_interval )
//			cout<<"��ǰʱ�䣺 "<<nt-_dt[0]<<"   rhomax: "<<_residual[0]/rhoref<<"   E: "<<_residual[1]/rhoE<<endl;
//
//		fout<<count<<"   "<<log(_residual[0]/rhoref)/log(10.0)<<endl;
//	}
//
//
//	hipEventDestroy(time_start);
//	hipEventDestroy(time_stop);
//
//	if ( log_history=='Y' )
//		fout.close();
//
//}

//void CCUDARkdgSolver::commuInfo() {
//	MPI_Request request;
//		
//	int *rho_buffer, *rhou_buffer, *rhov_buffer, *rhoE_buffer;
//	for (int i = 0; i < nprocs - 1; i++) {
//		int num = grid.local_innerBoundary_index[i].size();
//		rho_buffer = new int[num];
//		rhou_buffer = new int[num];
//		rhov_buffer = new int[num];
//		rhoE_buffer = new int[num];
//		for (int j = 0; j < num; j++) {
//			rho_buffer[j] = _cuarrays.freedom_rho[grid.local_innerBoundary_index[i].at(j)];
//			rhou_buffer[j] = _cuarrays.freedom_rhou[grid.local_innerBoundary_index[i].at(j)];
//			rhov_buffer[j] = _cuarrays.freedom_rhov[grid.local_innerBoundary_index[i].at(j)];
//			rhoE_buffer[j] = _cuarrays.freedom_rhoE[grid.local_innerBoundary_index[i].at(j)];
//		}
//		int dest = i < myid ? i : i + 1;
//		MPI_Isend(rho_buffer, num, MPI_DOUBLE, dest, 1 + 4 * num_commu, MPI_COMM_WORLD, &request);
//		MPI_Isend(rhou_buffer, num, MPI_DOUBLE, dest, 2 + 4 * num_commu, MPI_COMM_WORLD, &request);
//		MPI_Isend(rhov_buffer, num, MPI_DOUBLE, dest, 3 + 4 * num_commu, MPI_COMM_WORLD, &request);
//		MPI_Isend(rhoE_buffer, num, MPI_DOUBLE, dest, 4 + 4 * num_commu, MPI_COMM_WORLD, &request);
//			
//		MPI_Irecv(rho_buffer, num, MPI_DOUBLE, dest, 1 + 4 * num_commu, MPI_COMM_WORLD, &request);
//		MPI_Irecv(rhou_buffer, num, MPI_DOUBLE, dest, 2 + 4 * num_commu, MPI_COMM_WORLD, &request);
//		MPI_Irecv(rhov_buffer, num, MPI_DOUBLE, dest, 3 + 4 * num_commu, MPI_COMM_WORLD, &request);
//		MPI_Irecv(rhoE_buffer, num, MPI_DOUBLE, dest, 4 + 4 * num_commu++, MPI_COMM_WORLD, &request);	
//	}
//	MPI_Barrier(MPI_COMM_WORLD);
//	dealCommuData();
//}
//
//void CCUDARkdgSolver::dealCommuData() {
//	size_t pitch = _cuarrays.getDoublePitch();
//	for (int i = 0; i < nprocs - 1; i++) {
//		for (int j = 0; j < grid.local_innerBoundary_index[i].size(); j++) {
//			_freedom_rho[grid.local_innerBoundary_index[i].at(j)] =  rho_buffer[j];
//			_freedom_rhou[grid.local_innerBoundary_index[i].at(j)] = rhou_buffer[j];
//			_freedom_rhov[grid.local_innerBoundary_index[i].at(j)] = rhov_buffer[j];
//			_freedom_rhoE[grid.local_innerBoundary_index[i].at(j)] = rhoE_buffer[j];
//		}
//	}
//
//	hipMemcpy2DAsync(_cuarrays.freedom_rho, pitch, _freedom_rho, pitch, pitch, BASIS_FUNCTIONS, hipMemcpyHostToDevice);
//	hipMemcpy2DAsync(_cuarrays.freedom_rhou, pitch, _freedom_rhou, pitch, pitch, BASIS_FUNCTIONS, hipMemcpyHostToDevice);
//	hipMemcpy2DAsync(_cuarrays.freedom_rhov, pitch, _freedom_rhov, pitch, pitch, BASIS_FUNCTIONS, hipMemcpyHostToDevice);
//	hipMemcpy2DAsync(_cuarrays.freedom_rhoE, pitch, _freedom_rhoE, pitch, pitch, BASIS_FUNCTIONS, hipMemcpyHostToDevice);
//}

void CCUDARkdgSolver::outputSolution(double* result_rho, double* result_rhou, double* result_rhov, double* result_rhoE)
{
	/*for(int i = 0; i < 10; i++) {
		cout<<"output rhou:"<<result_rhou[i]<<endl;
	}*/
	ofstream fout(solution_file.c_str());
	cout<<"result file: "<<solution_file.c_str();
	if ( !fout )
	{
		cout<<"Failed to open solution file: "<<solution_file<<" and output will be omitted."<<endl;
		return;
	}

	int i;
	int vnum, tnum;
	double rho, u, v, rhoE, p, a, ma;
	vnum = grid.getVerticeNumber();
	tnum = grid.getTriangleNumber();

	fout<<"TITLE=RKDG"<<endl;
	fout<<"VARIABLES=X , Y , rho , u , v , p, Ma , FLAG"<<endl;
	fout<<"ZONE T= T1 N= "<<vnum<<" , E= "<<tnum<<" , ZONETYPE=FETRIANGLE"<<endl;
	fout<<"DATAPACKING=BLOCK"<<endl;
	fout<<"VARLOCATION=([1-2]=NODAL,[3-8]=CELLCENTERED)"<<endl;
	fout<<"DT=(SINGLE SINGLE SINGLE SINGLE SINGLE SINGLE SINGLE SINGLE)"<<endl;

	for ( i=0; i<vnum; ++i )
	{
		fout<<grid.vertice[i].getX()<<"  ";
		if ( i%6==0 )
		{
			fout<<endl;
		}
	}
	for ( i=0; i<vnum; ++i )
	{
		fout<<grid.vertice[i].getY()<<"  ";
		if ( i%6==0 )
		{
			fout<<endl;
		}
	}
	fout<<endl;
	for ( i=0; i<tnum; ++i )
	{
		fout<<result_rho[i]<<"  ";
		if ( i%6==0 )
		{
			fout<<endl;
		}
	}
	fout<<endl;
	for ( i=0; i<tnum; ++i )
	{
		fout<<result_rhou[i]/result_rho[i]<<"  ";
		if ( i%6==0 )
		{
			fout<<endl;
		}
	}
	fout<<endl;
	for ( i=0; i<tnum; ++i )
	{
		fout<<result_rhov[i]/result_rho[i]<<"  ";
		if ( i%6==0 )
		{
			fout<<endl;
		}
	}
	fout<<endl;
	for ( i=0; i<tnum; ++i )
	{
		rho = result_rho[i];
		u	= result_rhou[i]/rho;
		v	= result_rhov[i]/rho;
		rhoE = result_rhoE[i];

		p = (gamma-1)*(rhoE-0.5*rho*(u*u+v*v));

		fout<<p<<"  ";
		if ( i%6==0 )
		{
			fout<<endl;
		}
	}
	fout<<endl;
	for ( i=0; i<tnum; ++i )
	{
		rho = result_rho[i];
		u	= result_rhou[i]/rho;
		v	= result_rhov[i]/rho;
		rhoE = result_rhoE[i];

		p = (gamma-1)*(rhoE-0.5*rho*(u*u+v*v));
		a = sqrt(gamma*p/rho);
		if(i == 0) 
			cout<<"rho:"<<rho<<",u"<<u<<",v"<<v<<",rhoE"<<rhoE<<",p"<<p<<",a"<<a<<endl;
		ma = sqrt(u*u+v*v)/a;

		fout<<ma<<"  ";
		if ( i%6==0 )
		{
			fout<<endl;
		}
	}
	fout<<endl;
	// ���������
	for ( i=0; i<tnum; ++i )
	{
		fout<<"1"<<"  ";
		if ( i%6==0 )
		{
			fout<<endl;
		}
	}
	fout<<endl;

	for ( i=0; i<tnum; ++i )
	{
		fout<<grid.tri_vertice[3*i]+1<<"    "<<grid.tri_vertice[3*i+1]+1<<"    "<<grid.tri_vertice[3*i+2]+1<<endl;
	}
	fout.close();
}



void CCUDARkdgSolver::printConfig( ostream& out )
{
	if ( !out )
	{
		cerr<<"Invalid output stream and output will be omitted."<<endl;
		return;
	}

	// �����������
	out<<"===================="<<endl;
	out<<"Program configures: "<<endl;
	out<<"===================="<<endl;
	out<<"gamma:              "<<gamma<<endl;
	out<<"alpha:              "<<alpha*180/(4*atan(1.0))<<endl;
	out<<"mach:               "<<mach<<endl;
	out<<"cfl:                "<<cfl<<endl;
	out<<"rhoref:             "<<rhoref<<endl;
	out<<"pref:               "<<pref<<endl;
	out<<"time: 			  "<<_terminal_time<<endl;
	out<<"===================="<<endl;
	out<<"gridconf:           "<<gridconf<<endl;
	out<<"solution:           "<<solution_file<<endl;
	out<<"residualfile:       "<<residual_file<<endl;
	out<<"printinterval:      "<<print_interval<<endl;
	out<<"loghistory:         "<<log_history<<endl;
	out<<"===================="<<endl;
	out<<"threads_per_block:  "<<threads_per_block<<endl;
	out<<"reduction_threads:  "<<reduction_threads<<endl;
	out<<"===================="<<endl<<endl;
}


CCUDARkdgSolver::~CCUDARkdgSolver()
{
	delete []_freedom_rho;
	delete []_freedom_rhou;
	delete []_freedom_rhov;
	delete []_freedom_rhoE;

	hipHostFree(_residual);
	hipHostFree(_dt);
}